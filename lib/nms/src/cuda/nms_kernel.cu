#include "hip/hip_runtime.h"
// ------------------------------------------------------------------
// Faster R-CNN
// Copyright (c) 2015 Microsoft
// Licensed under The MIT License [see fast-rcnn/LICENSE for details]
// Written by Shaoqing Ren
// ------------------------------------------------------------------
#ifdef __cplusplus
extern "C" {
#endif

#include <math.h>
#include <stdio.h>
#include <float.h>
#include "nms_kernel.h"

#define CUDA_CHECK(condition) \
  /* Code block avoids redefinition of hipError_t error */ \
  do { \
    hipError_t error = condition; \
    if (error != hipSuccess) { \
      printf("%s\n", hipGetErrorString(error)); \
    } \
  } while (0)

#define DIVUP(m,n) ((m) / (n) + ((m) % (n) > 0))
int const threadsPerBlock = sizeof(unsigned long long) * 8;

__device__ inline float devIoU(float const * const a, float const * const b) {
  float left = fmaxf(a[0], b[0]), right = fminf(a[2], b[2]);
  float top = fmaxf(a[1], b[1]), bottom = fminf(a[3], b[3]);
  float width = fmaxf(right - left + 1, 0.f), height = fmaxf(bottom - top + 1, 0.f);
  float interS = width * height;
  float Sa = (a[2] - a[0] + 1) * (a[3] - a[1] + 1);
  float Sb = (b[2] - b[0] + 1) * (b[3] - b[1] + 1);
  return interS / (Sa + Sb - interS);
}

__global__ void nms_kernel(const int n_boxes, const float nms_overlap_thresh,
                           const float *dev_boxes, unsigned long long *dev_mask) {
  const int row_start = blockIdx.y;
  const int col_start = blockIdx.x;

  if (row_start > col_start) return;

  const int row_size =
        fminf(n_boxes - row_start * threadsPerBlock, threadsPerBlock);
  const int col_size =
        fminf(n_boxes - col_start * threadsPerBlock, threadsPerBlock);

  if (threadIdx.x < row_size) {
    const int cur_box_idx = threadsPerBlock * row_start + threadIdx.x;
    const float *cur_box = dev_boxes + cur_box_idx * 5;
    int start = 0;
    if (row_start == col_start) {
      start = threadIdx.x + 1;
    }
    unsigned long long t = 0;
    for (int i = start; i < col_size; i++) {
      const int box_col_idx = threadsPerBlock * col_start + i;
      if (devIoU(cur_box, dev_boxes + box_col_idx * 5) > nms_overlap_thresh) {
        t |= 1ULL << i;
      }
    }
    const int col_blocks = DIVUP(n_boxes, threadsPerBlock);
    dev_mask[cur_box_idx * col_blocks + col_start] = t;
  }
}


void _nms(long long *keep_out, long long *num_out,
          const int boxes_num, const float *boxes_dev, const float nms_overlap_thresh) {

  const int col_blocks = DIVUP(boxes_num, threadsPerBlock);

  unsigned long long* mask_dev = NULL;
  CUDA_CHECK(hipMalloc(&mask_dev,
                        boxes_num * col_blocks * sizeof(unsigned long long)));

  dim3 blocks(col_blocks, col_blocks);
  dim3 threads(threadsPerBlock);
  nms_kernel<<<blocks, threads>>>(boxes_num,
                                  nms_overlap_thresh,
                                  boxes_dev,
                                  mask_dev);

  unsigned long long *mask_host = new unsigned long long[boxes_num * col_blocks];
  CUDA_CHECK(hipMemcpy(mask_host,
                        mask_dev,
                        sizeof(unsigned long long) * boxes_num * col_blocks,
                        hipMemcpyDeviceToHost));

  unsigned long long *remv_boxes = new unsigned long long[col_blocks];
  memset(remv_boxes, 0, sizeof(unsigned long long) * col_blocks);

  long long &num_to_keep = *num_out;
  num_to_keep = 0;
  for (int i = 0; i < boxes_num; i++) {
    const int nblock = i / threadsPerBlock;
    const int inblock = i % threadsPerBlock;

    if (!(remv_boxes[nblock] & (1ULL << inblock))) {
      keep_out[num_to_keep++] = i;
      unsigned long long *p = mask_host + i * col_blocks;
      for (int j = nblock; j < col_blocks; j++) {
        remv_boxes[j] |= p[j];
      }
    }
  }

  CUDA_CHECK(hipFree(mask_dev));
  delete[] mask_host;
  delete[] remv_boxes;
}

#ifdef __cplusplus
}
#endif
